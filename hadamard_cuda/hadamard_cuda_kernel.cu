#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef FWT_KERNEL_CUH
#define FWT_KERNEL_CUH
#ifndef fwt_kernel_cuh
#define fwt_kernel_cuh

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_fp16.h>

#include "hip/hip_runtime_api.h"

namespace cg = cooperative_groups;

///////////////////////////////////////////////////////////////////////////////
// Elementary(for vectors less than elementary size) in-shared memory
// combined radix-2 + radix-4 Fast Walsh Transform
///////////////////////////////////////////////////////////////////////////////
#define ELEMENTARY_LOG2SIZE 11

__global__ void fwtBatch1Kernel(__half *d_Output, __half *d_Input, int log2N) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  const int N = 1 << log2N;
  const int base = blockIdx.x << log2N;

  //(2 ** 11) * 4 bytes == 8KB -- maximum s_data[] size for G80
  extern __shared__ __half s_data[];
  __half *d_Src = d_Input + base;
  __half *d_Dst = d_Output + base;

  for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
    s_data[pos] = d_Src[pos];
  }

  // Main radix-4 stages
  const int pos = threadIdx.x;

  for (int stride = N >> 2; stride > 0; stride >>= 2) {
    int lo = pos & (stride - 1);
    int i0 = ((pos - lo) << 2) + lo;
    int i1 = i0 + stride;
    int i2 = i1 + stride;
    int i3 = i2 + stride;

    cg::sync(cta);
    __half D0 = s_data[i0];
    __half D1 = s_data[i1];
    __half D2 = s_data[i2];
    __half D3 = s_data[i3];

    __half T;
    T = D0;
    D0 = __hadd(D0, D2);
    D2 = __hsub(T, D2);
    T = D1;
    D1 = __hadd(D1, D3);
    D3 = __hsub(T, D3);
    T = D0;
    s_data[i0] = __hadd(D0, D1);
    s_data[i1] = __hsub(T, D1);
    T = D2;
    s_data[i2] = __hadd(D2, D3);
    s_data[i3] = __hsub(T, D3);
  }

  // Do single radix-2 stage for odd power of two
  if (log2N & 1) {
    cg::sync(cta);

    for (int pos = threadIdx.x; pos < N / 2; pos += blockDim.x) {
      int i0 = pos << 1;
      int i1 = i0 + 1;

      __half D0 = s_data[i0];
      __half D1 = s_data[i1];
      s_data[i0] = __hadd(D0, D1);
      s_data[i1] = __hsub(D0, D1);
    }
  }

  cg::sync(cta);

  for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
    d_Dst[pos] = s_data[pos];
  }
}

////////////////////////////////////////////////////////////////////////////////
// Single in-global memory radix-4 Fast Walsh Transform pass
// (for strides exceeding elementary vector size)
////////////////////////////////////////////////////////////////////////////////
__global__ void fwtBatch2Kernel(__half *d_Output, __half *d_Input, int stride) {
  const int pos = blockIdx.x * blockDim.x + threadIdx.x;
  const int N = blockDim.x * gridDim.x * 4;

  __half *d_Src = d_Input + blockIdx.y * N;
  __half *d_Dst = d_Output + blockIdx.y * N;

  int lo = pos & (stride - 1);
  int i0 = ((pos - lo) << 2) + lo;
  int i1 = i0 + stride;
  int i2 = i1 + stride;
  int i3 = i2 + stride;

  __half D0 = d_Src[i0];
  __half D1 = d_Src[i1];
  __half D2 = d_Src[i2];
  __half D3 = d_Src[i3];

  __half T;
  T = D0;
  D0 = __hadd(D0, D2);
  D2 = __hsub(T, D2);
  T = D1;
  D1 = __hadd(D1, D3);
  D3 = __hsub(T, D3);
  T = D0;
  d_Dst[i0] = __hadd(D0, D1);
  d_Dst[i1] = __hsub(T, D1);
  T = D2;
  d_Dst[i2] = __hadd(D2, D3);
  d_Dst[i3] = __hsub(T, D3);
}

////////////////////////////////////////////////////////////////////////////////
// Put everything together: batched Fast Walsh Transform CPU front-end
////////////////////////////////////////////////////////////////////////////////
void fwtBatchGPU(__half *d_Data, int M, int log2N) {
  const int THREAD_N = 256;

  int N = 1 << log2N;
  dim3 grid((1 << log2N) / (4 * THREAD_N), M, 1);

  for (; log2N > ELEMENTARY_LOG2SIZE; log2N -= 2, N >>= 2, M <<= 2) {
    fwtBatch2Kernel<<<grid, THREAD_N>>>(d_Data, d_Data, N / 4);
    getLastCudaError("fwtBatch2Kernel() execution failed\n");
  }

  fwtBatch1Kernel<<<M, N / 4, N * sizeof(__half)>>>(d_Data, d_Data, log2N);
  getLastCudaError("fwtBatch1Kernel() execution failed\n");
}

////////////////////////////////////////////////////////////////////////////////
// Modulate two arrays
////////////////////////////////////////////////////////////////////////////////
__global__ void modulateKernel(__half *d_A, __half *d_B, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int numThreads = blockDim.x * gridDim.x;

  for (int pos = tid; pos < N; pos += numThreads) {
    d_A[pos] = __hmul(d_A[pos], __hdiv(d_B[pos], __int2half_rn(N)));
  }
}

// Interface to modulateKernel()
void modulateGPU(__half *d_A, __half *d_B, int N) {
  modulateKernel<<<128, 256>>>(d_A, d_B, N);
}

#endif
#endif
