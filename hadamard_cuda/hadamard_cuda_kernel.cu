#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef FWT_KERNEL_CUH
#define FWT_KERNEL_CUH
#ifndef fwt_kernel_cuh
#define fwt_kernel_cuh

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_fp16.h>
#include "hip/hip_runtime_api.h"

#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include <torch/types.h>


namespace cg = cooperative_groups;

///////////////////////////////////////////////////////////////////////////////
// Elementary(for vectors less than elementary size) in-shared memory
// combined radix-2 + radix-4 Fast Walsh Transform
///////////////////////////////////////////////////////////////////////////////
#define MAX_SMEM_LOG2SIZE 13


template <typename scalar_t>
__global__ static void fwtBatch1Kernel(scalar_t *d_Output, scalar_t *d_Input, int log2N) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  const int N = 1 << log2N;
  const int base = blockIdx.x << log2N;

  // 2 ** 13 bytes == 8KB -- maximum s_data[] size for A6000
  extern __shared__ __align__(8) unsigned char sdata_raw[];     // align to 8 bytes for double
  scalar_t *s_data = reinterpret_cast<scalar_t*>(sdata_raw);

  scalar_t *d_Src = d_Input + base;
  scalar_t *d_Dst = d_Output + base;

  for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
    s_data[pos] = d_Src[pos];
  }

  // Main radix-4 stages
  const int pos = threadIdx.x;

  for (int stride = N >> 2; stride > 0; stride >>= 2) {
    int lo = pos & (stride - 1);
    int i0 = ((pos - lo) << 2) + lo;
    int i1 = i0 + stride;
    int i2 = i1 + stride;
    int i3 = i2 + stride;

    cg::sync(cta);
    scalar_t D0 = s_data[i0];
    scalar_t D1 = s_data[i1];
    scalar_t D2 = s_data[i2];
    scalar_t D3 = s_data[i3];

    scalar_t T;
    T = D0;
    D0 = D0 + D2;
    D2 = T - D2;
    T = D1;
    D1 = D1 + D3;
    D3 = T - D3;
    T = D0;
    s_data[i0] = D0 + D1;
    s_data[i1] = T - D1;
    T = D2;
    s_data[i2] = D2 + D3;
    s_data[i3] = T - D3;
  }

  // Do single radix-2 stage for odd power of two
  if (log2N & 1) {
    cg::sync(cta);

    for (int pos = threadIdx.x; pos < N / 2; pos += blockDim.x) {
      int i0 = pos << 1;
      int i1 = i0 + 1;

      scalar_t D0 = s_data[i0];
      scalar_t D1 = s_data[i1];
      s_data[i0] = D0 + D1;
      s_data[i1] = D0 - D1;
    }
  }

  cg::sync(cta);

  for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
    d_Dst[pos] = s_data[pos];
  }
}


////////////////////////////////////////////////////////////////////////////////
// Single in-global memory radix-4 Fast Walsh Transform pass
// (for strides exceeding elementary vector size)
////////////////////////////////////////////////////////////////////////////////
template <typename scalar_t>
__global__ static void fwtBatch2Kernel(scalar_t *d_Output, scalar_t *d_Input, int stride) {
  const int pos = blockIdx.x * blockDim.x + threadIdx.x;
  const int N = blockDim.x * gridDim.x * 4;

  scalar_t *d_Src = d_Input + blockIdx.y * N;
  scalar_t *d_Dst = d_Output + blockIdx.y * N;

  int lo = pos & (stride - 1);
  int i0 = ((pos - lo) << 2) + lo;
  int i1 = i0 + stride;
  int i2 = i1 + stride;
  int i3 = i2 + stride;

  scalar_t D0 = d_Src[i0];
  scalar_t D1 = d_Src[i1];
  scalar_t D2 = d_Src[i2];
  scalar_t D3 = d_Src[i3];

  scalar_t T;
  T = D0;
  D0 = D0 + D2;
  D2 = T - D2;
  T = D1;
  D1 = D1 + D3;
  D3 = T - D3;

  T = D0;
  d_Dst[i0] = D0 + D1;
  d_Dst[i1] = T - D1;
  T = D2;
  d_Dst[i2] = D2 + D3;
  d_Dst[i3] = T - D3;
}


////////////////////////////////////////////////////////////////////////////////
// Single in-global memory radix-8 Fast Walsh Transform pass
// (for strides exceeding elementary vector size)
////////////////////////////////////////////////////////////////////////////////
template <typename scalar_t>
__global__ static void fwtBatch3Kernel(scalar_t *d_Output, scalar_t *d_Input, int stride) {
  const int pos = blockIdx.x * blockDim.x + threadIdx.x;
  const int N = blockDim.x * gridDim.x * 8;

  scalar_t *d_Src = d_Input + blockIdx.y * N;
  scalar_t *d_Dst = d_Output + blockIdx.y * N;

  int lo = pos & (stride - 1);
  int i0 = ((pos - lo) << 3) + lo;
  int i1 = i0 + stride;
  int i2 = i1 + stride;
  int i3 = i2 + stride;
  int i4 = i3 + stride;
  int i5 = i4 + stride;
  int i6 = i5 + stride;
  int i7 = i6 + stride;

  scalar_t D0 = d_Src[i0];
  scalar_t D1 = d_Src[i1];
  scalar_t D2 = d_Src[i2];
  scalar_t D3 = d_Src[i3];
  scalar_t D4 = d_Src[i4];
  scalar_t D5 = d_Src[i5];
  scalar_t D6 = d_Src[i6];
  scalar_t D7 = d_Src[i7];

  scalar_t T;
  T = D0;
  D0 = D0 + D4;
  D4 = T - D4;
  T = D1;
  D1 = D1 + D5;
  D5 = T - D5;
  T = D2;
  D2 = D2 + D6;
  D6 = T - D6;
  T = D3;
  D3 = D3 + D7;
  D7 = T - D7;

  T = D0;
  scalar_t E0 = D0 + D2;
  scalar_t E2 = T - D2;
  T = D1;
  scalar_t E1 = D1 + D3;
  scalar_t E3 = T - D3;
  T = D4;
  scalar_t E4 = D4 + D6;
  scalar_t E6 = T - D6;
  T = D5;
  scalar_t E5 = D5 + D7;
  scalar_t E7 = T - D7;

  T = E0;
  d_Dst[i0] = E0 + E1;
  d_Dst[i1] = T - E1;
  T = E2;
  d_Dst[i2] = E2 + E3;
  d_Dst[i3] = T - E3;
  T = E4;
  d_Dst[i4] = E4 + E5;
  d_Dst[i5] = T - E5;
  T = E6;
  d_Dst[i6] = E6 + E7;
  d_Dst[i7] = T - E7;
}


////////////////////////////////////////////////////////////////////////////////
// Put everything together: batched Fast Walsh Transform CPU front-end
////////////////////////////////////////////////////////////////////////////////
__host__ extern void fwtBatchGPU(torch::Tensor& d_Data, size_t M, int log2N) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(d_Data.scalar_type(), "fwtBatchGPU", [&] {
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    int scalar_type_log2size = log2(sizeof(scalar_t));
    scalar_t *data_ptr = d_Data.data_ptr<scalar_t>();

    int N = 1 << log2N;
    const int THREAD_N = 256;
    dim3 grid(N / (8 * THREAD_N), M, 1);

    for (; log2N > MAX_SMEM_LOG2SIZE + scalar_type_log2size; log2N -= 3, N >>= 3, M <<= 3) {
      fwtBatch3Kernel<<<grid, THREAD_N, 0, stream>>>(data_ptr, data_ptr, N / 8);
      getLastCudaError("fwtBatch2Kernel() execution failed\n");
    }

    fwtBatch1Kernel<<<M, N / 4, N * sizeof(scalar_t), stream>>>(data_ptr, data_ptr, log2N);
    getLastCudaError("fwtBatch1Kernel() execution failed\n");
  });
}


////////////////////////////////////////////////////////////////////////////////
// Modulate two arrays
////////////////////////////////////////////////////////////////////////////////
template <typename scalar_t>
__global__ static void modulateKernel(scalar_t *d_A, scalar_t *d_B, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int numThreads = blockDim.x * gridDim.x;

  for (int pos = tid; pos < N; pos += numThreads) {
    d_A[pos] = __hmul(d_A[pos], __hdiv(d_B[pos], __int2half_rn(N)));
  }
}


// Interface to modulateKernel()
template <typename scalar_t>
__host__ extern void modulateGPU(scalar_t *d_A, scalar_t *d_B, int N) {
  modulateKernel<<<128, 256>>>(d_A, d_B, N);
}

#endif
#endif
